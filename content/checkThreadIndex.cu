#include "bits/stdc++.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>
#define in std::cin
#define out std::cout
#define rep(i,N) for(LL i=0;i<N;++i)
typedef long long int LL;

#define CHECK(call)	\
{	\
	const hipError_t error = call;	\
	if (error != hipSuccess)	\
	{	\
		out << "Error: " << __FILE__ << ":" << __LINE__ << ", ";	\
		out << "code:" << error << ", reason: " << hipGetErrorString(error) << std::endl;	\
		exit(1);	\
	}	\
}	\

void printMatrix(int *C, const int nx, const int ny)
{
	int *ic = C;
	out << "\nMatrix: (" << nx << "." << ny << ")\n";
	rep(iy, ny)
	{
		rep(ix, nx)
		{
			out << ic[ix];
		}
		ic += nx;
		out << "\n";
	}
	out << "\n";
}

__global__ void printThreadIndex(int *A, const int nx, const int ny)
{
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned int idx = iy * nx + ix;
	printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) global index %d ival %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
}

int main(int argc, char **argv)
{
	out << argv[0] << " Starting...\n";

	// デバイス情報を取得
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	out << "Using Device " << dev << ": " << deviceProp.name << "\n";
	CHECK(hipSetDevice(dev));

	// 行列の次元を設定
	int nx = 8, ny = 6;
	int nxy = nx * ny;
	int nBytes = nxy * sizeof(float);

	// ホストメモリを確保
	int *h_A = (int *)malloc(nBytes);

	// ホスト行列を整数で初期化
	rep(i, nxy) h_A[i] = i;
	printMatrix(h_A, nx, ny);

	// デバイスメモリを確保
	int *d_MatA;
	CHECK(hipMalloc((void **)&d_MatA, nBytes));

	// ホストからデバイスへデータを転送
	CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));

	// 実行設定をセットアップ
	dim3 block(4, 2);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

	// カーネルを呼び出す
	printThreadIndex<<< grid, block >>>(d_MatA, nx, ny);
	CHECK(hipDeviceSynchronize());

	// ホストとデバイスのメモリを解放
	CHECK(hipFree(d_MatA));
	free(h_A);

	// デバイスをリセット
	CHECK(hipDeviceReset());
}
